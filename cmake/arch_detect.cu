
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int count = 0;
    if (hipGetDeviceCount(&count) != hipSuccess) return -1;
    if (count == 0) return -1;
    for (int device = 0; device < count; ++device) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, device) == hipSuccess) {
            printf("%d%d;", prop.major, prop.minor);
        }
    }
    return 0;
}
