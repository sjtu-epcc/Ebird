#include "hip/hip_runtime.h"
/*
 * Created Date: Sunday, May 5th 2019, 1:49:54 pm
 * Author: Raphael-Hao
 * -----
 * Last Modified: Friday, March 13th 2020, 4:32:36 pm
 * Modified By: Raphael-Hao
 * -----
 * Copyright (c) 2019 Happy
 *
 * Were It to Benefit My Country, I Would Lay Down My Life !
 */
#include <hip/hip_runtime.h>
#include <stream_singleton.h>
#include <tensor.h>

namespace ebird {

__global__ void tensor_sum_fkernel(float *ptr1, float *ptr2, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    ptr1[i] = ptr1[i] + ptr2[i];
  }
  __syncthreads();
}

__global__ void tensor_sum_dkernel(double *ptr1, double *ptr2, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    ptr1[i] = ptr1[i] + ptr2[i];
  }
  __syncthreads();
}

template <class value_type>
void tensor_sum(value_type *ptr1, value_type *ptr2, int size,
                hipStream_t &cur_stream) {
  if (sizeof(value_type) == 2) {
    fprintf(stderr,
            "HALF precision not supported so far!@softmax_layer.cu:line27\n");
    exit(1);
  } else if (sizeof(value_type) == 4) {
    // single precision
    tensor_sum_fkernel<<<(size + 255) / 256, 256, 0, cur_stream>>>(
        (float *)ptr1, (float *)ptr2, size);
  } else {
    // double precision
    tensor_sum_dkernel<<<(size + 255) / 256, 256, 0, cur_stream>>>(
        (double *)ptr1, (double *)ptr2, size);
  }
}

__global__ void tensor_copy_fkernel(float *ptr1, float *ptr2, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    ptr2[i] = ptr1[i];
  }
  __syncthreads();
}

__global__ void tensor_copy_dkernel(double *ptr1, double *ptr2, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    ptr2[i] = ptr1[i];
  }
  __syncthreads();
}

template <class value_type>
void tensor_copy(value_type *ptr1, value_type *ptr2, int size,
                 hipStream_t &cur_stream) {
  if (sizeof(value_type) == 2) {
    fprintf(stderr,
            "HALF precision not supported so far!@softmax_layer.cu:line27\n");
    exit(1);
  } else if (sizeof(value_type) == 4) {
    // single precision
    tensor_copy_fkernel<<<(size + 255) / 256, 256, 0, cur_stream>>>(
        (float *)ptr1, (float *)ptr2, size);
  } else {
    // double precision
    tensor_copy_dkernel<<<(size + 255) / 256, 256, 0, cur_stream>>>(
        (double *)ptr1, (double *)ptr2, size);
  }
}

__global__ void tensor_scale_fkernel(float *ptr1, float s, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    ptr1[i] = ptr1[i] * s;
  }
  __syncthreads();
}

__global__ void tensor_scale_dkernel(double *ptr1, double s, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    ptr1[i] = ptr1[i] * s;
  }
  __syncthreads();
}

template <class value_type>
void tensor_scale(value_type *ptr1, value_type s, int size,
                  hipStream_t &cur_stream) {
  if (sizeof(value_type) == 2) {
    fprintf(stderr,
            "HALF precision not supported so far!@softmax_layer.cu:line27\n");
    exit(1);
  } else if (sizeof(value_type) == 4) {
    // single precision
    tensor_scale_fkernel<<<(size + 255) / 256, 256, 0, cur_stream>>>(
        (float *)ptr1, (float)s, size);
  } else {
    // double precision
    tensor_scale_dkernel<<<(size + 255) / 256, 256, 0, cur_stream>>>(
        (double *)ptr1, (double)s, size);
  }
}

template void tensor_scale<float>(float *ptr1, float s, int size,
                                  hipStream_t &cur_stream);
template void tensor_scale<double>(double *ptr1, double s, int size,
                                   hipStream_t &cur_stream);
template void tensor_sum<float>(float *ptr1, float *ptr2, int size,
                                hipStream_t &cur_stream);
template void tensor_sum<double>(double *ptr1, double *ptr2, int size,
                                 hipStream_t &cur_stream);
template void tensor_copy<float>(float *ptr1, float *ptr2, int size,
                                 hipStream_t &cur_stream);
template void tensor_copy<double>(double *ptr1, double *ptr2, int size,
                                  hipStream_t &cur_stream);

}  // namespace ebird
